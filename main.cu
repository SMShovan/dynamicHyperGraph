#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <cmath>
#include <cstdlib>
#include <ctime>
#include <climits>

//Function to create a random 2-D vector
std::vector<std::vector<int>> createRandom2DVector(int n, int m, int r1, int r2) {
    std::vector<std::vector<int>> vec2d(n);
    std::srand(std::time(0)); // Seed for random number generation

    for (int i = 0; i < n; ++i) {
        int innerSize = rand() % m + 1; // Random inner size from 1 to m
        vec2d[i].resize(innerSize);
        for (int j = 0; j < innerSize; ++j) {
            vec2d[i][j] = rand() % (r2 - r1 + 1) + r1; // Random value in range [r1, r2]
        }
    }

    return vec2d;
}

int nextMultipleOf32(int num) {
    return ((num + 32) / 32) * 32;
}

int nextMultipleOf4(int num) {
    return ((num + 4) / 4) * 4;
}

std::pair<std::vector<int>, std::vector<int>> flatten2DVector(const std::vector<std::vector<int>>& vec2d) {
    std::vector<int> vec1d;
    std::vector<int> vec2dto1d(vec2d.size());

    int index = 0;
    for (size_t i = 0; i < vec2d.size(); ++i) {
        vec2dto1d[i] = index;
        int innerSize = vec2d[i].size();
        int paddedSize = nextMultipleOf4(innerSize);
        for (int j = 0; j < paddedSize; ++j) {
            if (j < innerSize) {
                vec1d.push_back(vec2d[i][j]);
            } else if (j == paddedSize - 1) {
                vec1d.push_back(INT_MIN); // Padding with negative infinity
            } else {
                vec1d.push_back(0); // Padding with zeros
            }
            ++index;
        }
    }

    return {vec1d, vec2dto1d};
}

void print2DVector(const std::vector<std::vector<int>>& vec2d) {
    std::cout << "2D Vector (Matrix Form):" << std::endl;
    for (const auto& row : vec2d) {
        for (int val : row) {
            std::cout << val << " ";
        }
        std::cout << std::endl;
    }
}

void printVector(const std::vector<int>& vec, const std::string& name) {
    std::cout << name << ": [ ";
    for (int val : vec) {
        std::cout << val << " ";
    }
    std::cout << "]" << std::endl;
}

void checkCuda(hipError_t result) {
    if (result != hipSuccess) {
        std::cerr << "CUDA Runtime Error: " << hipGetErrorString(result) << std::endl;
        exit(-1);
    }
}


__device__ int ceil_log2(int x) {
    int log = 0;
    while ((1 << log) < x) ++log;
    return log;
}
__device__ int floor_log2(int x) {
    int log = 0;
    while (x >>= 1) ++log;
    return log;
}

// Structure for Red-Black Tree Node
struct RBTreeNode {
    int index;
    int value;
    int length;
    int size;
    bool color; // Red or Black
    RBTreeNode* left;
    RBTreeNode* right;
    RBTreeNode* parent;
};

// Kernel to build an empty binary tree
__global__ void buildEmptyBinaryTree(RBTreeNode* nodes, int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < n) {
        nodes[tid].index = tid;
        nodes[tid].left = (2 * tid + 1 < n) ? &nodes[2 * tid + 1] : nullptr;
        nodes[tid].right = (2 * tid + 2 < n) ? &nodes[2 * tid + 2] : nullptr;
        nodes[tid].parent = (tid == 0) ? nullptr : &nodes[(tid - 1) / 2];
    }
}

// Kernel to store items into internal nodes
__global__ void storeItemsIntoNodes(RBTreeNode* nodes, int* indices, int* values, int n, int totalSize) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < n) {
        int log2_tid = floor_log2(tid + 1);
        int log2_n = floor_log2(n);
        int index =  ((2 * (tid + 1  - (1<<log2_tid))) + 1) * (1 << log2_n) / (1 << log2_tid);
        int index2 = min(index, index - (index/2) + (n + 1 - (1<< log2_n)));
        index2--;


        // # if __CUDA_ARCH__>=200
        //     printf("tid is %d \n", tid + 1);
        //     printf("J(i) is %d \n", (tid + 1  - (1<<log2_tid)));
        //     printf("log2_n is %d \n", log2_n);
        //     printf("index is %d \n", index);
        //     printf("size is %d \n", n);
        // #endif

        nodes[tid].size = totalSize;
        if (index2 < n) {
            nodes[tid].index = indices[index2];
            nodes[tid].value = values[index2];
            if (index2 < n - 1) {
                nodes[tid].length = values[index2 + 1] - values[index2];
            } else {
                nodes[tid].length = totalSize - values[index2];
            }
        }
    }
}

// Kernel to color the nodes red or black
__global__ void colorNodes(RBTreeNode* nodes, int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < n) {
        nodes[tid].color = (tid % 2 == 0); // Simplified coloring: alternating red (false) and black (true)
    }
}


// Kernel to print each node from the device
__global__ void printEachNode(RBTreeNode* nodes, int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid <= n) {
        RBTreeNode* current = nodes;
        while (current != nullptr && current->index != tid) {
            if (current->index > tid) {
                current = current->left;
            } else {
                current = current->right;
            }
        }
        if (current != nullptr) {
            printf("Node %d: Index = %d, Value = %d, Length = %d, Size = %d, Color = %s\n",
                   tid, current->index, current->value, current->length, current->size, current->color ? "Black" : "Red");
        }
    }
}
// Kernel to find and print nodes in the tree
__global__ void findNode(RBTreeNode* nodes, int* searchIndices, int searchSize) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < searchSize) {
        int searchIndex = searchIndices[tid];
        RBTreeNode* current = nodes;
        while (current != nullptr && current->index != searchIndex) {
            if (current->index > searchIndex) {
                current = current->left;
            } else {
                current = current->right;
            }
        }
        if (current != nullptr) {
            printf("Node %d: Index = %d, Value = %d, Length = %d, Color = %s\n",
                   searchIndex, current->index, current->value, current->length, current->color ? "Black" : "Red");
        } else {
            printf("Node %d: Not Found\n", searchIndex);
        }
    }
}

__global__ void insertNode(RBTreeNode* nodes, int* flatValues, int* insertIndices, int* insertValues, int* insertSizes, int insertSize, int* partialSolution) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < insertSize) {
        int insertIndex = insertIndices[tid];
        int* values;
        int numValues; 
        if (tid == 0){
            values = insertValues;
            numValues = insertSizes[tid];
        }
        else{
            values = insertValues + insertSizes[tid - 1];
            numValues = insertSizes[tid] - insertSizes[tid - 1];
        }
        // Search for the node by index
        RBTreeNode* current = nodes;
        while (current != nullptr && current->index != insertIndex) {
            if (current->index > insertIndex) {
                current = current->left;
            } else {
                current = current->right;
            }
        }

        // If node is found
        if (current != nullptr) {
            int valueIndex = current->value;
            
            // Navigate flatValues array to find the position to insert
            for (int i = 0; i < numValues; ++i) {
                bool isOverflow = false;
                while (flatValues[valueIndex] != 0 && flatValues[valueIndex] != INT_MIN) {
                    if (flatValues[valueIndex + 1] == INT_MIN)
                    {
                        # if __CUDA_ARCH__>=200
                            printf("Overflow of thread %d: position %d start %d of size %d \n", tid, valueIndex + 1, i, numValues - i);
                            partialSolution[tid * 3] = valueIndex + 1;
                            partialSolution[tid * 3 + 1] = i; 
                            partialSolution[tid * 3 + 2] = numValues - i; 
                        #endif
                        isOverflow = true;
                    }
                    if (isOverflow)
                    {
                        break;
                    }
                    valueIndex++;
                }
                // Insert the new value
                if (isOverflow)
                    break;
                if (flatValues[valueIndex] != INT_MIN)
                    flatValues[valueIndex] = values[i];
            }

            // Update the node's value to the new index
            current->value = valueIndex;
        }
    }
}

void constructRedBlackTree(int* h_indices, int* h_values, int n, int* flatValues, int flatValuesSize) {
    const int fixedSize = 1024; // Fixed size for d_flatValues

    // Check if fixedSize is at least flatValuesSize
    if (fixedSize < flatValuesSize) {
        std::cerr << "Overflow: fixedSize is less than flatValuesSize" << std::endl;
        return;
    }

    RBTreeNode* d_nodes;
    int* d_indices;
    int* d_values;
    int* d_flatValues;
    int* d_insertIndices;
    int* d_insertValues;
    int* d_insertSizes;
    int* d_partialSolution;

    // Allocate device memory
    checkCuda(hipMalloc(&d_nodes, n * sizeof(RBTreeNode)));
    checkCuda(hipMalloc(&d_indices, n * sizeof(int)));
    checkCuda(hipMalloc(&d_values, n * sizeof(int)));

    // Allocate fixed memory for d_flatValues
    checkCuda(hipMalloc(&d_flatValues, fixedSize * sizeof(int)));

    // Copy first portion from flatValues
    checkCuda(hipMemcpy(d_flatValues, flatValues, flatValuesSize * sizeof(int), hipMemcpyHostToDevice));

    // Initialize remaining portion to zero
    checkCuda(hipMemset(d_flatValues + flatValuesSize, 0, (fixedSize - flatValuesSize) * sizeof(int)));

    checkCuda(hipMalloc(&d_insertIndices, n * sizeof(int)));
    checkCuda(hipMalloc(&d_insertValues, n * 3 * sizeof(int)));  // Allocate max size for values
    checkCuda(hipMalloc(&d_insertSizes, n * sizeof(int)));
    checkCuda(hipMalloc(&d_partialSolution, 3 * n * sizeof(int)));

    checkCuda(hipMemcpy(d_indices, h_indices, n * sizeof(int), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(d_values, h_values, n * sizeof(int), hipMemcpyHostToDevice));

    // Copy dummy insert indices and values for initial tree construction
    checkCuda(hipMemcpy(d_insertIndices, h_indices, n * sizeof(int), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(d_insertValues, h_values, n * sizeof(int), hipMemcpyHostToDevice));

    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;

    // Step 1: Build the empty binary tree
    buildEmptyBinaryTree<<<numBlocks, blockSize>>>(d_nodes, n);
    checkCuda(hipDeviceSynchronize());

    // Step 2: Store items into internal nodes
    storeItemsIntoNodes<<<numBlocks, blockSize>>>(d_nodes, d_indices, d_values, n, flatValuesSize);
    checkCuda(hipDeviceSynchronize());

    // Step 3: Color the nodes
    colorNodes<<<numBlocks, blockSize>>>(d_nodes, n);
    checkCuda(hipDeviceSynchronize());

    // Print each node from the device
    std::cout << "Printing the tree from the device:" << std::endl;
    printEachNode<<<numBlocks, blockSize>>>(d_nodes, n);
    checkCuda(hipDeviceSynchronize());

    // Prepare data for insertion
    std::vector<std::pair<int, std::vector<int>>> insertVector = {{2, {200 }}, {4, {400, 300, 310, 320, 330, 340, 350}}, {6, {600, 700, 650}}};
    std::vector<int> insertIndices(insertVector.size());
    std::vector<int> insertValues;
    std::vector<int> insertSizes(insertVector.size());
    std::vector<int> partialSolution(insertVector.size() * 3, 0);
    
    int count = 0;
    for (size_t i = 0; i < insertVector.size(); ++i) {
        insertIndices[i] = insertVector[i].first;
        insertValues.insert(insertValues.end(), insertVector[i].second.begin(), insertVector[i].second.end());
        if (i == 0)
            insertSizes[i] = insertVector[i].second.size();
        else 
            insertSizes[i] = insertSizes[i-1] + insertVector[i].second.size();
    }

    checkCuda(hipMemcpy(d_insertIndices, insertIndices.data(), insertIndices.size() * sizeof(int), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(d_insertValues, insertValues.data(), insertValues.size() * sizeof(int), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(d_insertSizes, insertSizes.data(), insertSizes.size() * sizeof(int), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(d_partialSolution, partialSolution.data(), insertSizes.size() * sizeof(int) * 3, hipMemcpyHostToDevice));

    // Insert nodes into the Red-Black Tree
    insertNode<<<(insertIndices.size() + blockSize - 1) / blockSize, blockSize>>>(d_nodes, d_flatValues, d_insertIndices, d_insertValues, d_insertSizes, insertIndices.size(), d_partialSolution);
    checkCuda(hipDeviceSynchronize());

    // Copy flat values back to host and print them
    std::vector<int> updatedFlatValues(fixedSize);
    checkCuda(hipMemcpy(updatedFlatValues.data(), d_flatValues, fixedSize * sizeof(int), hipMemcpyDeviceToHost));

    checkCuda(hipMemcpy(partialSolution.data(), d_partialSolution, insertSizes.size() * sizeof(int) * 3, hipMemcpyDeviceToHost));
    printVector(partialSolution, "Partial solution");
    printVector(updatedFlatValues, "Updated Flattened Values (vec1d)");

    // Free device memory
    checkCuda(hipFree(d_insertIndices));
    checkCuda(hipFree(d_insertValues));
    checkCuda(hipFree(d_insertSizes));
    checkCuda(hipFree(d_indices));
    checkCuda(hipFree(d_values));
    checkCuda(hipFree(d_nodes));
    checkCuda(hipFree(d_flatValues));
}

int main() {
    int n = 8;
    std::vector<std::vector<int>> random2DVec = createRandom2DVector(n, 5, 1, 100);

    print2DVector(random2DVec);

    // Flatten the 2D vector
    auto flattened = flatten2DVector(random2DVec);
    std::vector<int> flatValues = flattened.first;
    std::vector<int> flatIndices = flattened.second;

    // Print the flattened vectors
    printVector(flatValues, "Flattened Values (vec1d)");
    printVector(flatIndices, "Flattened Indices (vec2dto1d)");

    int* h_values = flatIndices.data();
    int* h_indices = new int[flatIndices.size()];
    for (size_t i = 0; i < flatIndices.size(); ++i) {
        h_indices[i] = i + 1;
    }

    constructRedBlackTree(h_indices, h_values, n, flatValues.data(), flatValues.size());


    delete[] h_indices;
    return 0;
}