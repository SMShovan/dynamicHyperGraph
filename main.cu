#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <cmath>
#include <cstdlib>
#include <ctime>
#include <climits>

//Function to create a random 2-D vector
std::vector<std::vector<int>> createRandom2DVector(int n, int m, int r1, int r2) {
    std::vector<std::vector<int>> vec2d(n);
    std::srand(std::time(0)); // Seed for random number generation

    for (int i = 0; i < n; ++i) {
        int innerSize = rand() % m + 1; // Random inner size from 1 to m
        vec2d[i].resize(innerSize);
        for (int j = 0; j < innerSize; ++j) {
            vec2d[i][j] = rand() % (r2 - r1 + 1) + r1; // Random value in range [r1, r2]
        }
    }

    return vec2d;
}

int nextMultipleOf32(int num) {
    return ((num + 31) / 32) * 32;
}

std::pair<std::vector<int>, std::vector<int>> flatten2DVector(const std::vector<std::vector<int>>& vec2d) {
    std::vector<int> vec1d;
    std::vector<int> vec2dto1d(vec2d.size());

    int index = 0;
    for (size_t i = 0; i < vec2d.size(); ++i) {
        vec2dto1d[i] = index;
        int innerSize = vec2d[i].size();
        int paddedSize = nextMultipleOf32(innerSize);
        for (int j = 0; j < paddedSize; ++j) {
            if (j < innerSize) {
                vec1d.push_back(vec2d[i][j]);
            } else if (j == paddedSize - 1) {
                vec1d.push_back(INT_MIN); // Padding with negative infinity
            } else {
                vec1d.push_back(0); // Padding with zeros
            }
            ++index;
        }
    }

    return {vec1d, vec2dto1d};
}

void print2DVector(const std::vector<std::vector<int>>& vec2d) {
    std::cout << "2D Vector (Matrix Form):" << std::endl;
    for (const auto& row : vec2d) {
        for (int val : row) {
            std::cout << val << " ";
        }
        std::cout << std::endl;
    }
}

void printVector(const std::vector<int>& vec, const std::string& name) {
    std::cout << name << ": [ ";
    for (int val : vec) {
        std::cout << val << " ";
    }
    std::cout << "]" << std::endl;
}

void checkCuda(hipError_t result) {
    if (result != hipSuccess) {
        std::cerr << "CUDA Runtime Error: " << hipGetErrorString(result) << std::endl;
        exit(-1);
    }
}


__device__ int ceil_log2(int x) {
    int log = 0;
    while ((1 << log) < x) ++log;
    return log;
}
__device__ int floor_log2(int x) {
    int log = 0;
    while (x >>= 1) ++log;
    return log;
}

// Structure for Red-Black Tree Node
struct RBTreeNode {
    int index;
    int value;
    int length;
    bool color; // Red or Black
    RBTreeNode* left;
    RBTreeNode* right;
    RBTreeNode* parent;
};

// Kernel to build an empty binary tree
__global__ void buildEmptyBinaryTree(RBTreeNode* nodes, int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < n) {
        nodes[tid].index = tid;
        nodes[tid].left = (2 * tid + 1 < n) ? &nodes[2 * tid + 1] : nullptr;
        nodes[tid].right = (2 * tid + 2 < n) ? &nodes[2 * tid + 2] : nullptr;
        nodes[tid].parent = (tid == 0) ? nullptr : &nodes[(tid - 1) / 2];
    }
}

// Kernel to store items into internal nodes
__global__ void storeItemsIntoNodes(RBTreeNode* nodes, int* indices, int* values, int n, int totalSize) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < n) {
        int log2_tid = floor_log2(tid + 1);
        int log2_n = floor_log2(n);
        int index =  ((2 * (tid + 1  - (1<<log2_tid))) + 1) * (1 << log2_n) / (1 << log2_tid);
        int index2 = min(index, index - (index/2) + (n + 1 - (1<< log2_n)));
        index2--;


        // # if __CUDA_ARCH__>=200
        //     printf("tid is %d \n", tid + 1);
        //     printf("J(i) is %d \n", (tid + 1  - (1<<log2_tid)));
        //     printf("log2_n is %d \n", log2_n);
        //     printf("index is %d \n", index);
        //     printf("size is %d \n", n);
        // #endif


        if (index2 < n) {
            nodes[tid].index = indices[index2];
            nodes[tid].value = values[index2];
            if (index2 < n - 1) {
                nodes[tid].length = values[index2 + 1] - values[index2];
            } else {
                nodes[tid].length = totalSize - values[index2];
            }
        }
    }
}

// Kernel to color the nodes red or black
__global__ void colorNodes(RBTreeNode* nodes, int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < n) {
        nodes[tid].color = (tid % 2 == 0); // Simplified coloring: alternating red (false) and black (true)
    }
}


// Kernel to print each node from the device
__global__ void printEachNode(RBTreeNode* nodes, int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid <= n) {
        RBTreeNode* current = nodes;
        while (current != nullptr && current->index != tid) {
            if (current->index > tid) {
                current = current->left;
            } else {
                current = current->right;
            }
        }
        if (current != nullptr) {
            printf("Node %d: Index = %d, Value = %d, Length = %d, Color = %s\n",
                   tid, current->index, current->value, current->length, current->color ? "Black" : "Red");
        }
    }
}
// Kernel to find and print nodes in the tree
__global__ void findNode(RBTreeNode* nodes, int* searchIndices, int searchSize) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < searchSize) {
        int searchIndex = searchIndices[tid];
        RBTreeNode* current = nodes;
        while (current != nullptr && current->index != searchIndex) {
            if (current->index > searchIndex) {
                current = current->left;
            } else {
                current = current->right;
            }
        }
        if (current != nullptr) {
            printf("Node %d: Index = %d, Value = %d, Length = %d, Color = %s\n",
                   searchIndex, current->index, current->value, current->length, current->color ? "Black" : "Red");
        } else {
            printf("Node %d: Not Found\n", searchIndex);
        }
    }
}

// Kernel to insert nodes in the tree
__global__ void insertNode(RBTreeNode* nodes, int* flatValues, int* insertIndices, int* insertValues, int insertSize) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < insertSize) {
        int insertIndex = insertIndices[tid];
        int insertValue = insertValues[tid];

        // Search for the node by index
        RBTreeNode* current = nodes;
        while (current != nullptr && current->index != insertIndex) {
            if (current->index > insertIndex) {
                current = current->left;
            } else {
                current = current->right;
            }
        }

        // If node is found
        if (current != nullptr) {
            int valueIndex = current->value;
            
            // Navigate flatValues array to find the position to insert
            while (flatValues[valueIndex] != 0) {
                valueIndex++;
            }
            
            // Insert the new value
            flatValues[valueIndex] = insertValue;

            // Update the node's value to the new index
            current->value = valueIndex;
        }
    }
}

void constructRedBlackTree(int* h_indices, int* h_values, int n, int* flatValues, int flatValuesSize) {
    RBTreeNode* d_nodes;
    int* d_indices;
    int* d_values;
    int* d_flatValues;
    int* d_insertIndices;
    int* d_insertValues;

    checkCuda(hipMalloc(&d_nodes, n * sizeof(RBTreeNode)));
    checkCuda(hipMalloc(&d_indices, n * sizeof(int)));
    checkCuda(hipMalloc(&d_values, n * sizeof(int)));
    checkCuda(hipMalloc(&d_flatValues, flatValuesSize * sizeof(int)));
    checkCuda(hipMalloc(&d_insertIndices, n * sizeof(int)));
    checkCuda(hipMalloc(&d_insertValues, n * sizeof(int)));

    checkCuda(hipMemcpy(d_indices, h_indices, n * sizeof(int), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(d_values, h_values, n * sizeof(int), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(d_flatValues, flatValues, flatValuesSize * sizeof(int), hipMemcpyHostToDevice));
    // Copy dummy insert indices and values for initial tree construction
    checkCuda(hipMemcpy(d_insertIndices, h_indices, n * sizeof(int), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(d_insertValues, h_values, n * sizeof(int), hipMemcpyHostToDevice));

    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;

    // Step 1: Build the empty binary tree
    buildEmptyBinaryTree<<<numBlocks, blockSize>>>(d_nodes, n);
    checkCuda(hipDeviceSynchronize());

    // Step 2: Store items into internal nodes
    storeItemsIntoNodes<<<numBlocks, blockSize>>>(d_nodes, d_indices, d_values, n, flatValuesSize);
    checkCuda(hipDeviceSynchronize());

    // Step 3: Color the nodes
    colorNodes<<<numBlocks, blockSize>>>(d_nodes, n);
    checkCuda(hipDeviceSynchronize());

    // Print each node from the device
    std::cout << "Printing the tree from the device:" << std::endl;
    printEachNode<<<numBlocks, blockSize>>>(d_nodes, n);
    checkCuda(hipDeviceSynchronize());

    // Prepare data for insertion
    std::vector<std::pair<int, int>> insertVector = {{2, 200}, {4, 400}, {6, 600}};
    std::vector<int> insertIndices(insertVector.size());
    std::vector<int> insertValues(insertVector.size());
    for (size_t i = 0; i < insertVector.size(); ++i) {
        insertIndices[i] = insertVector[i].first;
        insertValues[i] = insertVector[i].second;
    }

    checkCuda(hipMemcpy(d_insertIndices, insertIndices.data(), insertIndices.size() * sizeof(int), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(d_insertValues, insertValues.data(), insertValues.size() * sizeof(int), hipMemcpyHostToDevice));

    // Insert nodes into the Red-Black Tree
    insertNode<<<(insertIndices.size() + blockSize - 1) / blockSize, blockSize>>>(d_nodes, d_flatValues, d_insertIndices, d_insertValues, insertIndices.size());
    checkCuda(hipDeviceSynchronize());

    checkCuda(hipMemcpy(flatValues, d_flatValues, flatValuesSize * sizeof(int), hipMemcpyDeviceToHost));

    printVector(std::vector<int>(flatValues, flatValues + flatValuesSize), "Updated Flattened Values (vec1d)");


    // Free device memory
    checkCuda(hipFree(d_insertIndices));
    checkCuda(hipFree(d_insertValues));
    checkCuda(hipFree(d_indices));
    checkCuda(hipFree(d_values));
    checkCuda(hipFree(d_nodes));
    checkCuda(hipFree(d_flatValues));
}

int main() {
    int n = 8;
    std::vector<std::vector<int>> random2DVec = createRandom2DVector(n, 5, 1, 100);

    print2DVector(random2DVec);

    // Flatten the 2D vector
    auto flattened = flatten2DVector(random2DVec);
    std::vector<int> flatValues = flattened.first;
    std::vector<int> flatIndices = flattened.second;

    // Print the flattened vectors
    printVector(flatValues, "Flattened Values (vec1d)");
    printVector(flatIndices, "Flattened Indices (vec2dto1d)");

    int* h_values = flatIndices.data();
    int* h_indices = new int[flatIndices.size()];
    for (size_t i = 0; i < flatIndices.size(); ++i) {
        h_indices[i] = i + 1;
    }

    constructRedBlackTree(h_indices, h_values, n, flatValues.data(), flatValues.size());

    

    delete[] h_indices;
    return 0;
}